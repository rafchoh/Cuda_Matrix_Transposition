#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#define MAT_SIZE 1024
#define BLOCK_COLS 64
#define BLOCK_ROWS 8
#define PADDING_VALUE 0

__global__ void transposeMatrix(int* output_data, const int* input_data, int width, int padding) {
    __shared__ int tile[BLOCK_COLS][BLOCK_COLS];

    int x = blockIdx.x * BLOCK_COLS + threadIdx.x;
    int y = blockIdx.y * BLOCK_COLS + threadIdx.y;
    int index_in = y * width + x;

#pragma unroll
    for (int j = 0; j < BLOCK_COLS; j += BLOCK_ROWS) {
        tile[threadIdx.y + j][threadIdx.x] = (x < width && (y + j) < width) ? input_data[index_in + j * width] : padding;
    }

    __syncthreads();

    x = blockIdx.y * BLOCK_COLS + threadIdx.x;
    y = blockIdx.x * BLOCK_COLS + threadIdx.y;
    int index_out = y * width + x;

#pragma unroll
    for (int j = 0; j < BLOCK_COLS; j += BLOCK_ROWS) {
        output_data[index_out + j * width] = tile[threadIdx.x][threadIdx.y + j];
    }
}

void printMatrix(const int* matrix, int width, int height) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            std::cout << matrix[i * width + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    const dim3 blockSize(BLOCK_COLS, BLOCK_ROWS);
    const dim3 gridSize(MAT_SIZE / BLOCK_COLS, MAT_SIZE / BLOCK_COLS);

    int* h_inputMatrix = new int[MAT_SIZE * MAT_SIZE];
    int* h_transMatrix = new int[MAT_SIZE * MAT_SIZE];

    srand(static_cast<unsigned>(time(nullptr)));
    for (int i = 0; i < MAT_SIZE * MAT_SIZE; ++i) {
        h_inputMatrix[i] = rand() % 101; 
    }

    int* d_inputMatrix, * d_transMatrix;
    hipMalloc((void**)&d_inputMatrix, MAT_SIZE * MAT_SIZE * sizeof(int));
    hipMalloc((void**)&d_transMatrix, MAT_SIZE * MAT_SIZE * sizeof(int));

    hipMemcpy(d_inputMatrix, h_inputMatrix, MAT_SIZE * MAT_SIZE * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    transposeMatrix << <gridSize, blockSize >> > (d_transMatrix, d_inputMatrix, MAT_SIZE, PADDING_VALUE);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_transMatrix, d_transMatrix, MAT_SIZE * MAT_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Input Matrix:" << std::endl;
    //printMatrix(h_inputMatrix, MAT_SIZE, MAT_SIZE);

    std::cout << "\n \n";

    std::cout << "\nTransposed Matrix:" << std::endl;
   // printMatrix(h_transMatrix, MAT_SIZE, MAT_SIZE);

    std::cout << "\nExecution Time is: " << milliseconds << " ms" << std::endl;

    delete[] h_inputMatrix;
    delete[] h_transMatrix;
    hipDeviceReset();
    hipFree(d_inputMatrix);
    hipFree(d_transMatrix);

    return 0;
}
